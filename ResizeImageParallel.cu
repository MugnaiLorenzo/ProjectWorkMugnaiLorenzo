#include "hip/hip_runtime.h"
#include "ResizeImageParallel.cuh"
#include <hip/hip_runtime.h>
#include <iostream>

__constant__ float d_kernel_const[5 * 5];

__device__ float clamp(float val) {
    return val < 0 ? 0 : (val > 255 ? 255 : val);
}

// Kernel non vettoriale: 1 thread per canale
__global__ void convolveKernel_NonVec(unsigned char* input, unsigned char* output,
                                      int width, int height, int ksize) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int c = threadIdx.z;  // canale RGB

    int offset = ksize / 2;
    if (x >= width || y >= height || c >= 3) return;

    float acc = 0.0f;
    for (int i = -offset; i <= offset; ++i) {
        for (int j = -offset; j <= offset; ++j) {
            int nx = x + j;
            int ny = y + i;
            if (nx >= 0 && nx < width && ny >= 0 && ny < height) {
                int idx = (ny * width + nx) * 3 + c;
                acc += input[idx] * d_kernel_const[(i + offset) * ksize + (j + offset)];
            }
        }
    }
    output[(y * width + x) * 3 + c] = static_cast<unsigned char>(clamp(acc));
}

// Kernel vettoriale: 1 thread per pixel, elabora tutti e 3 i canali
__global__ void convolveKernel_Vec(unsigned char* input, unsigned char* output,
                                   int width, int height, int ksize) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    int offset = ksize / 2;
    if (x >= width || y >= height) return;

    for (int c = 0; c < 3; ++c) {
        float acc = 0.0f;
        for (int i = -offset; i <= offset; ++i) {
            for (int j = -offset; j <= offset; ++j) {
                int nx = x + j;
                int ny = y + i;
                if (nx >= 0 && nx < width && ny >= 0 && ny < height) {
                    int idx = (ny * width + nx) * 3 + c;
                    acc += input[idx] * d_kernel_const[(i + offset) * ksize + (j + offset)];
                }
            }
        }
        output[(y * width + x) * 3 + c] = static_cast<unsigned char>(clamp(acc));
    }
}

unsigned char* applyConvolutionCUDA(const unsigned char* input, int width, int height,
                                    const std::vector<std::vector<float>>& kernel2D,
                                    ConvolutionType type,
                                    int threadsX, int threadsY) {
    int imgSize = width * height * 3;
    int ksize = static_cast<int>(kernel2D.size());
    int klen = ksize * ksize;

    std::vector<float> kernel1D(klen);
    for (int i = 0; i < ksize; ++i)
        for (int j = 0; j < ksize; ++j)
            kernel1D[i * ksize + j] = kernel2D[i][j];

    unsigned char* d_input;
    unsigned char* d_output;

    hipMalloc(&d_input, imgSize);
    hipMalloc(&d_output, imgSize);

    hipMemcpy(d_input, input, imgSize, hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(d_kernel_const), kernel1D.data(), sizeof(float) * klen);

    dim3 block(threadsX, threadsY);
    dim3 grid((width + threadsX - 1) / threadsX, (height + threadsY - 1) / threadsY);

    if (type == NonVectorized) {
        dim3 block3D(threadsX, threadsY, 3);  // un thread per canale RGB
        convolveKernel_NonVec<<<grid, block3D>>>(d_input, d_output, width, height, ksize);
    } else {
        convolveKernel_Vec<<<grid, block>>>(d_input, d_output, width, height, ksize);
    }

    unsigned char* result = new unsigned char[imgSize];
    hipMemcpy(result, d_output, imgSize, hipMemcpyDeviceToHost);

    hipFree(d_input);
    hipFree(d_output);

    return result;
}
